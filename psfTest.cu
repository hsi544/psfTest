#include "hip/hip_runtime.h"
#include <iostream>
#include "complex_gpu.h"

__global__ void subtract_psf_kernel(hipFloatComplex *d_subnfftsp,
                                    const hipFloatComplex __restrict__ *d_psf,
                                    int numWinTs,
                                    int subnfftspSize,
                                    int offset,
                                    const int *d_i1,
                                    const int *d_i2,
                                    const int *d_i3,
                                    const int *d_i4,
                                    const int *d_k1,
                                    const int *d_k2,
                                    const int *d_k3,
                                    const int *d_k4,
                                    int sizePSF1,
                                    int sizePSF2,
                                    int sizePSF3,
                                    hipFloatComplex *nfftval
                                   )
{

   int id = threadIdx.x + blockDim.x*blockIdx.x;

   if( id < numWinTs*subnfftspSize)
   {
      int thisWinTs = id/subnfftspSize;

      int p1 = d_k1[id] - d_i1[thisWinTs];
      int p2 = d_k2[id] - d_i2[thisWinTs];
      int p3 = d_k3[id] - d_i3[thisWinTs];
      int p4 = d_k4[id] - d_i4[thisWinTs];

      int idxOut = p4*sizePSF3*sizePSF2*sizePSF1+p3*sizePSF2*sizePSF1+p2*sizePSF1+p1+offset;

      d_subnfftsp[id] = d_subnfftsp[id] - d_psf[idxOut]*nfftval[thisWinTs];
   }

}

int main()
{
#if 0
   const int niter = 1000;
   const int numWinTs = 192;
   const int p1_size =  45;
   const int p2_size = 45;
   const int p3_size = 17;
   const int p4_size = 1;
   const int k1_size =  23;
   const int k2_size = 23;
   const int k3_size = 9;
   const int k4_size = 1;
   const int subnfftspSize = 4761;
   const int psfOffset = 17212;
   dim3 threads4(256,1,1);
   dim3 blocks4;
   blocks4.x = (numWinTs*subnfftspSize)/threads4.x;
   blocks4.x +=((numWinTs*subnfftspSize)%threads4.x)?1:0;

   hipFloatComplex *d_vecIn;
   hipFloatComplex *d_vecPSF;
   hipMalloc(&d_vecIn, subnfftspSize*sizeof(hipFloatComplex));
   cudaCheckError();
   hipMalloc(&d_vecPSF, psize*sizeof(hipFloatComplex));
   cudaCheckError();
   // copy data to GPU
   hipMemcpy(d_vecIn, &vecIn[0], ksize*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
   cudaCheckError();
   hipMemcpy(d_vecPSF, &vecPSF[0], psize*sizeof(hipFloatComplex), hipMemcpyHostToDevice);
   cudaCheckError();
   hipFloatComplex d_nfftval = make_hipFloatComplex(crealf(nfftval), cimag(nfftval));

   for(int iter = 0 ; iter < niter ; iter++)
   {
   
      subtract_psf_kernel<<<blocks4, threads4>>>(hipFloatComplex *d_subnfftsp,
                                                 const hipFloatComplex __restrict__ *d_psf,
                                                 numWinTs,
                                                 subnfftspSize,
                                                 psfOffset,
                                                 const int *d_i1,
                                                 const int *d_i2,
                                                 const int *d_i3,
                                                 const int *d_i4,
                                                 const int *d_k1,
                                                 const int *d_k2,
                                                 const int *d_k3,
                                                 const int *d_k4,
                                                 p1_size,
                                                 p2_size,
                                                 p3_size,
                                                 hipFloatComplex *nfftval);
   }
#endif
   return 0;
}
